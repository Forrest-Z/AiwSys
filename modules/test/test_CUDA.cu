#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

__global__ void add(float * x, float *y, float * z, int n){
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        int stride = blockDim.x * gridDim.x;
        
        for (int i = index; i < n; i += stride){
                z[i] = x[i] + y[i];
        }
}

int main(){

        int dev = 0;
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, dev);
        std::cout << "使用GPU device " << dev << ": " << devProp.name << std::endl;
        std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
        std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
        std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
        std::cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "每个EM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;


        int N = 1 << 20;
        int nBytes = N * sizeof (float);
        float *x, *y, *z;
        x = (float*)malloc(nBytes);
        y = (float*)malloc(nBytes);
        z = (float*)malloc(nBytes);

        for (int i = 0; i < N; i++){
                x[i] = 10.0;
                y[i] = 20.0;
        }

        float *d_x, *d_y, *d_z;
        hipMalloc((void**)&d_x, nBytes);
        hipMalloc((void**)&d_y, nBytes);
        hipMalloc((void**)&d_z, nBytes);

        hipMemcpy((void*)d_x, (void*)x, nBytes, hipMemcpyHostToDevice);
        hipMemcpy((void*)d_y, (void*)y, nBytes, hipMemcpyHostToDevice);
        
        dim3 blockSize(256);
        // 4096
        dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
        
        add << < gridSize, blockSize >> >(d_x, d_y, d_z, N);

        hipMemcpy((void*)z, (void*)d_z, nBytes, hipMemcpyDeviceToHost);

        float maxError = 0.0;
        for (int i = 0; i < N; i++){
                maxError = fmax(maxError, (float)(fabs(z[i] - 30.0)));
        }
        printf ("test max default: %.4f\n", maxError);

        hipFree(d_x);
        hipFree(d_y);
        hipFree(d_z);
        free(x);
        free(y);
        free(z);

        return 0;
}